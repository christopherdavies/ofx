
#include <hip/hip_runtime.h>
__device__ float Pq2Luma(float N) {
  float pq_m1 = 0.1593017578125; // ( 2610.0 / 4096.0 ) / 4.0;
  float pq_m2 = 78.84375; // ( 2523.0 / 4096.0 ) * 128.0;
  float pq_c1 = 0.8359375; // 3424.0 / 4096.0 or pq_c3 - pq_c2 + 1.0;
  float pq_c2 = 18.8515625; // ( 2413.0 / 4096.0 ) * 32.0;
  float pq_c3 = 18.6875; // ( 2392.0 / 4096.0 ) * 32.0;
  float pq_C = 10000.0;

  float Np = powf( N, 1.0 / pq_m2 );
  float L = Np - pq_c1;
  if ( L < 0.0 ) {
    L = 0.0;
  }
  L = L / ( pq_c2 - pq_c3 * Np );
  L = powf( L, 1.0 / pq_m1 );
  L = L * pq_C;
  return L; // returns cd/m^2
}

__device__ float Luma2Pq(float C) {
  float pq_m1 = 0.1593017578125; // ( 2610.0 / 4096.0 ) / 4.0;
  float pq_m2 = 78.84375; // ( 2523.0 / 4096.0 ) * 128.0;
  float pq_c1 = 0.8359375; // 3424.0 / 4096.0 or pq_c3 - pq_c2 + 1.0;
  float pq_c2 = 18.8515625; // ( 2413.0 / 4096.0 ) * 32.0;
  float pq_c3 = 18.6875; // ( 2392.0 / 4096.0 ) * 32.0;
  float pq_C = 10000.0;

  float L = C / pq_C;
  float Lm = powf( L, pq_m1 );
  float N = ( pq_c1 + pq_c2 * Lm ) / ( 1.0 + pq_c3 * Lm );
  N = powf( N, pq_m2 );
  return N;
}

__device__ float Luma(float R, float G, float B) {
  float lumaRec2020 = R * 0.2627f + G * 0.6780f + B * 0.0593f;
  return lumaRec2020;
} 


__global__ void LegalOverlayKernel(int p_Width, int p_Height, float p_Luminance, float p_OverlayR, float p_OverlayG, float p_OverlayB, int p_OverlayDisplay, int p_OverlayLuma, const float* p_Input, float* p_Output)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
       const int index = ((y * p_Width) + x) * 4;
      
      float t = Luma2Pq(p_Luminance);
      int overlay = p_OverlayDisplay;
      int lumaWarn = p_OverlayLuma;

      float rOver, gOver, bOver;
      rOver = p_OverlayR / 100;
      gOver = p_OverlayG / 100;
      bOver = p_OverlayB / 100;

      float r, g, b, a;
      r = p_Input[index + 0];
      g = p_Input[index + 1];
      b = p_Input[index + 2];
      a = p_Input[index + 3];

      if  (lumaWarn && overlay) {
        float rY, gY, bY;
        rY = Pq2Luma(r);
        gY = Pq2Luma(g);
        bY = Pq2Luma(b);
        float luma = Luma(rY, gY, bY);
        float pqLuma = Luma2Pq(luma);
        if ( pqLuma >= t ) {
          // if any channel is over threshold, replace with overlay color
          r = rOver;
          g = gOver;
          b = bOver;
        }
      } else if (overlay && ( r >= t || g >= t || b >= t ) ) {
        r = rOver;
        g = gOver;
        b = bOver;
      }

      p_Output[index + 0] = r;
      p_Output[index + 1] = g;
      p_Output[index + 2] = b;
      p_Output[index + 3] = a;   
   }
}

void RunCudaKernel(int p_Width, int p_Height, float p_Luminance, float* p_OverlayRgb, int p_OverlayDisplay, int p_OverlayLuma, const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

    LegalOverlayKernel<<<blocks, threads>>>(p_Width, p_Height, p_Luminance, p_OverlayRgb[0], p_OverlayRgb[1], p_OverlayRgb[2], p_OverlayDisplay, p_OverlayLuma, p_Input, p_Output);
}


